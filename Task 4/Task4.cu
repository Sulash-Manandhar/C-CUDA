#include "lodepng.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

unsigned int devWidth;
unsigned char getRed(unsigned char *image, unsigned int row, unsigned int col)
{
    unsigned int i = (row * devWidth * 4) + (col * 4);
    return image[i];
}
unsigned char getGreen(unsigned char *image, unsigned int row,
                                  unsigned int col)
{
    unsigned int i = (row * devWidth * 4) + (col * 4) + 1;
    return image[i];
}
unsigned char getBlue(unsigned char *image, unsigned int row, unsigned int col)
{
    unsigned int i = (row * devWidth * 4) + (col * 4) + 2;
    return image[i];
}
unsigned char getAlpha(unsigned char *image, unsigned int row,
                                  unsigned int col)
{
    unsigned int i = (row * devWidth * 4) + (col * 4) + 3;
    return image[i];
}
void setRed(unsigned char *image, unsigned int row, unsigned int col,
                       unsigned char red)
{
    unsigned int i = (row * devWidth * 4) + (col * 4);
    image[i] = red;
}
void setGreen(unsigned char *image, unsigned int row, unsigned int col,
                         unsigned char green)
{
    unsigned int i = (row * devWidth * 4) + (col * 4) + 1;
    image[i] = green;
}
void setBlue(unsigned char *image, unsigned int row, unsigned int col,
                        unsigned char blue)
{
    unsigned int i = (row * devWidth * 4) + (col * 4) + 2;
    image[i] = blue;
}
void setAlpha(unsigned char *image, unsigned int row, unsigned int col,
                         unsigned char alpha)
{
    unsigned int i = (row * devWidth * 4) + (col * 4) + 3;
    image[i] = alpha;
}
void changeImage(unsigned char *image, unsigned char *newImage,
                            unsigned int *width)
{
    int row = blockIdx.x + 1;
    int col = threadIdx.x + 1;
    devWidth = *width;
    unsigned redTL, redTC, redTR;
    unsigned redL, redC, redR;
    unsigned redBL, redBC, redBR;
    unsigned newRed;
    unsigned greenTL, greenTC, greenTR;
    unsigned greenL, greenC, greenR;
    unsigned greenBL, greenBC, greenBR;
    unsigned newGreen;
    unsigned blueTL, blueTC, blueTR;
    unsigned blueL, blueC, blueR;
    unsigned blueBL, blueBC, blueBR;
    unsigned newBlue;
    setGreen(newImage, row, col, getGreen(image, row, col));
    setBlue(newImage, row, col, getBlue(image, row, col));
    setAlpha(newImage, row, col, 255);
    redTL = getRed(image, row - 1, col - 1);
    redTC = getRed(image, row - 1, col);
    redTR = getRed(image, row - 1, col + 1);
    redL = getRed(image, row, col - 1);
    redC = getRed(image, row, col);
    redR = getRed(image, row, col + 1);
    redBL = getRed(image, row + 1, col - 1);
    redBC = getRed(image, row + 1, col);
    redBR = getRed(image, row + 1, col + 1);
    newRed = (redTL + redTC + redTR + redL + redC + redR + redBL + redBC + redBR) / 9;
    setRed(newImage, row, col, newRed);
    greenTL = getGreen(image, row - 1, col - 1);
    greenTC = getGreen(image, row - 1, col);
    greenTR = getGreen(image, row - 1, col + 1);
    greenL = getGreen(image, row, col - 1);
    greenC = getGreen(image, row, col);
    greenR = getGreen(image, row, col + 1);
    greenBL = getGreen(image, row + 1, col - 1);
    greenBC = getGreen(image, row + 1, col);
    greenBR = getGreen(image, row + 1, col + 1);
    newGreen =
        (greenTL + greenTC + greenTR + greenL + greenC + greenR + greenBL + greenBC + greenBR) / 9;
    setGreen(newImage, row, col, newGreen);
    blueTL = getBlue(image, row - 1, col - 1);
    blueTC = getBlue(image, row - 1, col);
    blueTR = getBlue(image, row - 1, col + 1);
    blueL = getBlue(image, row, col - 1);
    blueC = getBlue(image, row, col);
    blueR = getBlue(image, row, col + 1);
    blueBL = getBlue(image, row + 1, col - 1);
    blueBC = getBlue(image, row + 1, col);
    blueBR = getBlue(image, row + 1, col + 1);
    newBlue =
        (blueTL + blueTC + blueTR + blueL + blueC + blueR + blueBL + blueBC + blueBR) / 9;
    setBlue(newImage, row, col, newBlue);
}
int main(int argc, char **argv)
{
    unsigned char *image;
    const char *filename = argv[1];
    const char *newFileName = "blurred.png";
    unsigned char *newImage;
    unsigned int height = 0, width = 0;
    lodepng_decode32_file(&image, &width, &height, filename);
    newImage = (unsigned char *)malloc(height * width * 4 * sizeof(unsigned char));
    unsigned char *gpuImage;
    hipMalloc((void **)&gpuImage, sizeof(char) * height * width * 4);
    hipMemcpy(gpuImage, image, sizeof(char) * height * width * 4,
               hipMemcpyHostToDevice);
    unsigned char *gpuNewImage;
    hipMalloc((void **)&gpuNewImage, sizeof(char) * height * width * 4);
    unsigned int *gpuWidth;
    hipMalloc((void **)&gpuWidth, sizeof(int));
    hipMemcpy(gpuWidth, &width, sizeof(int), hipMemcpyHostToDevice);
    printf("Image width = %d height = %d\n", width, height);
    changeImage<<<height - 1, width - 1>>>(gpuImage, gpuNewImage, gpuWidth);
    hipDeviceSynchronize();
    hipMemcpy(newImage, gpuNewImage, sizeof(char) * height * width * 4,
               hipMemcpyDeviceToHost);
    lodepng_encode32_file(newFileName, newImage, width, height);
    return 0;
}
