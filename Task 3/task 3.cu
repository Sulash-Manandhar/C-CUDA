#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>


__device__ int pw_check(char *attempt) {
	char pw1[] = "YD58";
	char pw2[] = "CD25";
	char pw3[] = "DF14";
	char pw4[] = "GF45";


	char *e = attempt;
	char *a = attempt;
	char *g = attempt;
	char *l = attempt;
	char *p1 = pw1;
	char *p2 = pw2;
	char *p3 = pw3;
	char *p4 = pw4;

	while(*e == *p1) { 
		if(*e == '\0') 
		{
			printf("Password: %s\n",pw1);
			break;
		}

		e++;
		p1++;
	}
	
	while(*a == *p2) { 
		if(*a == '\0') 
		{
			printf("Password: %s\n",pw2);
			break;
		}

		a++;
		p2++;
	}

	while(*g == *p3) { 
		if(*g == '\0') 
		{
			printf("Password: %s\n",pw3);
			break;
		}

		g++;
		p3++;
	}

	while(*l == *p4) { 
		if(*l == '\0') 
		{
			printf("Password: %s\n",pw4);
			return 1;
		}

		l++;
		p4++;
	}
	return 0;

}

__global__ void  kernel() {
	char k1,k2;

	char pw[7];
	pw[6] = '\0';

	int i = blockIdx.x+65;
	int j = threadIdx.x+65;
	char firstMatch = i; 
	char secondMatch = j; 

	pw[0] = firstMatch;
	pw[1] = secondMatch;
	for(k1='0'; k1<='9'; k1++){
		for(k2='0'; k2<='9'; k2++){
			
				
					pw[2] = k1;
					pw[3] = k2;
					 
					if(pw_check(pw)) {
					} 
					else {
	     			//printf("tried: %s\n", pw);		  
					}
				
			
		}
	}
}

int time_diff(struct timespec *start, 
	struct timespec *finish, 
	long long int *diff) {
	long long int dsec =  finish->tv_sec - start->tv_sec; 
	long long int dnanosec =  finish->tv_nsec - start->tv_nsec; 
	if(dnanosec < 0 ) {
		dsec--;
		dnanosec += 1000000000; 
	} 
	*diff = dsec * 1000000000 + dnanosec;
	return !(*diff > 0);
}


int main() {

	struct  timespec start, finish;
	long long int time_elapsed;
	clock_gettime(CLOCK_MONOTONIC, &start);

	kernel <<<26,26>>>();
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &finish);
	time_diff(&start, &finish, &time_elapsed);
	printf("Time elapsed was %lldnanosec or %0.9lfsec\n", time_elapsed, (time_elapsed/1.0e9)); 

	return 0;
}


